#include <hip/hip_runtime.h>

#define THRESHOLD 10010000
#define WARPSIZE  32
#define SHAREDMEMSIZE 10144

__device__ void bubbleSort(int pixelsToSort[512], int length){
    for(int i = 0; i < length; i++ )
    {
        for(int j = 0; j < length-1; j++)
        {
            if( pixelsToSort[j] > pixelsToSort[j+1]){
                int tmp = pixelsToSort[j];
                pixelsToSort[j] = pixelsToSort[j+1];
                pixelsToSort[j+1] = tmp;
            }
        }
    }
}

__device__ int cudaGetFirstNotInColor(int *image, int x, int row, int imageWidth, int color){
    for (int i = x; i < imageWidth; ++i)
    {
        if(THRESHOLD < (color - image[row*imageWidth + i])){
            return i;
        }
    }
    return -1;
}

__device__ int cudaGetNextInColor(int *image, int x, int row, int imageWidth, int color){
    for (int i = x + 1; i < imageWidth; ++i)
    {
        if(THRESHOLD >= (color - image[row*imageWidth + i])){
            return i-1;
        }
    }
    return imageWidth - 1;
}

__global__ void sortRows(int *image, int imageHeight, int imageWidth, int colorMode){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < imageHeight)
    {
        int startingX = 0;
        int finishX = 0;

        while(finishX < imageWidth)
        {
            startingX = cudaGetFirstNotInColor(image, startingX, row, imageWidth, colorMode);
            finishX = cudaGetNextInColor(image, startingX, row, imageWidth, colorMode);

            if(startingX < 0)
                break;

            int pixelsToSortLength = finishX - startingX;
            int *pixelsToSort = new int[pixelsToSortLength];
            for (int i = 0; i < pixelsToSortLength; ++i)
            {
                pixelsToSort[i] = image[row*imageWidth + startingX + i];
            }

            bubbleSort(pixelsToSort, pixelsToSortLength);

            for (int i = 0; i < pixelsToSortLength; ++i)
            {
                image[row*imageWidth + startingX + i] = pixelsToSort[i];
            }

            startingX = finishX + 1;

            free(pixelsToSort);
        }
    }
}

__global__ void optimizedSortRows(int *image, int imageHeight, int imageWidth, int colorMode){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < imageHeight)
    {
        int startingX = 0;
        int finishX = 0;
        int *pixelsToSort = new int[1024];
        while(finishX < imageWidth)
        {
            startingX = cudaGetFirstNotInColor(image, startingX, row, imageWidth, colorMode);
            finishX = cudaGetNextInColor(image, startingX, row, imageWidth, colorMode);

            if(startingX < 0)
                break;

            int pixelsToSortLength = (finishX - startingX < 1024) ? finishX - startingX : 1024;

            for (int i = 0; i < pixelsToSortLength; ++i)
            {
                pixelsToSort[i] = image[row*imageWidth + startingX + i];
            }

            for(int i = 0; i < pixelsToSortLength; i++ )
            {
                for(int j = 0; j < pixelsToSortLength-1; j++)
                {
                    if( pixelsToSort[j] > pixelsToSort[j+1]){
                        pixelsToSort[j] = pixelsToSort[j] ^ pixelsToSort[j+1];
                        pixelsToSort[j+1] = pixelsToSort[j] ^ pixelsToSort[j+1];
                        pixelsToSort[j] = pixelsToSort[j] ^ pixelsToSort[j+1];
                    }
                }
            }

            for (int i = 0; i < pixelsToSortLength; ++i)
            {
                image[row*imageWidth + startingX + i] = pixelsToSort[i];
            }

            startingX = finishX + 1;
        }

        free(pixelsToSort);
    }
}